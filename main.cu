// C++ implementation of the approach

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

// Function to print the intersection
void findIntersection(int intervals[][2], int N)
{
	// First interval
	int l = intervals[0][0];
	int r = intervals[0][1];

	// Check rest of the intervals and find the intersection
	for (int i = 1; i < N; i++) {

		// If no intersection exists
		if (intervals[i][0] > r || intervals[i][1] < l) {
			cout << -1;
			return;
		}

		// Else update the intersection
		else {
			l = max(l, intervals[i][0]);
			r = min(r, intervals[i][1]);
		}
	}

	cout << "[" << l << ", " << r << "]";
}

// Driver code
int main()
{
	int intervals[][2] = {
		{ 1, 6 },
		{ 2, 8 },
		{ 3, 10 },
		{ 5, 8 }
	};
	int N = sizeof(intervals) / sizeof(intervals[0]);
	findIntersection(intervals, N);
}

/*
for i en el largo de A
    for j in el largo de B

    Si A[i] se intersecta con B[j]
        Guardar (i,j)
*/
