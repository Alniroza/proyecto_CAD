#include <iostream>
#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

void Read(int** A, int** B, int *la, int *lb, const char *filename) {    
	FILE *fp;
	fp = fopen(filename, "r");
  	fscanf(fp, "%d %d\n", la, lb);

	int* Atemp = new int[(*la) * 2];
	int* Btemp = new int[(*lb) * 2];

	for (int i = 0; i < (*la); i++){
		fscanf(fp, "%d %d\n", &(Atemp[2*i]), &(Atemp[2*i + 1]));
	}

	for (int j = 0; j < (*lb); j++){
		fscanf(fp, "%d %d\n", &(Btemp[2*j]), &(Btemp[2*j + 1]));
	}

	*A = Atemp;
	*B = Btemp;
}

void ReadSoA(int** A, int** B, int *la, int *lb, const char *filename) {    
	FILE *fp;
	fp = fopen(filename, "r");
  	fscanf(fp, "%d %d\n", la, lb);

	int* Atemp = new int[(*la) * 2];
	int* Btemp = new int[(*lb) * 2];

	for (int i = 0; i < (*la); i++){
		fscanf(fp, "%d %d\n", &(Atemp[i]), &(Atemp[*la + i]));
	}

	for (int j = 0; j < (*lb); j++){
		fscanf(fp, "%d %d\n", &(Btemp[j]), &(Btemp[*lb + j]));
	}

	*A = Atemp;
	*B = Btemp;
}

void Write(int* intersecciones, int la, int lb, const char *filename) {
	FILE *fp;
	fp = fopen(filename, "w");

	for (int i = 0; i < (la*lb*2); i++){
		if (i%2 == 0){
			if ((i != 0) && (intersecciones[i] == 0) && (intersecciones[i+1] == 0)){
				break;
			}
			fprintf(fp, "%d %d\n", intersecciones[i], intersecciones[i + 1]);
		}
	}
	fclose(fp);
}

bool seIntersecta(int aStart, int aEnd, int bStart, int bEnd){

	if ((aEnd < bStart) || (bEnd < aStart)){
		return false;
	}
	else{
		return true;
	}
}

/*
for i en el largo de A
    for j in el largo de B

    Si A[i] se intersecta con B[j]
        Guardar (i,j)
*/
void interseccionConjuntos(int* A, int *B,int *intersecciones, 
						int la, int lb){		
	
	int aStart, aEnd, bStart, bEnd;
	int posicion = 0;
	
	for (int i = 0; i < la; i++){
		aStart = A[2*i];
		aEnd = A[2*i + 1];

		for (int j = 0; j < lb; j++){
			bStart = B[2*j];
			bEnd = B[2*j + 1];

			if (seIntersecta(aStart, aEnd, bStart, bEnd)){
				//Guardo el número del intervalo (partiendo de 0)
				printf("%d ", posicion);
				intersecciones[posicion] = i;
				intersecciones[posicion + 1] = j;
				posicion += 2;
			}
		}
	}
}

int main(int argc, char **argv){

	// Largo del arreglo A y B, respectivamente.
	int la, lb;
	// Conjuntos de intervalos A y B.
	int *A, *B;
	int *intersecciones;
	clock_t t1, t2;
	
	char filename[] = {"input.txt\0"};
	char outputFilename[] = {"output.txt\0"};

	Read(&A, &B, &la, &lb, filename); 

	// for (int i = 0; i < la*2; i++){
	// 	std::cout << A[i] << std::endl;
	// }

	// Parte CPU

	intersecciones = new int[la*lb*2];

	t1 = clock();
	interseccionConjuntos(A, B, intersecciones, la, lb);
	t2 = clock();

	double ms = 1000.0 * (double)(t2 -t1) / CLOCKS_PER_SEC;

	std::cout << "Tiempo algoritmo en CPU = " << ms << "[ms]" << std::endl;

	Write(intersecciones, la, lb, outputFilename);

	delete[] intersecciones;

	return 0;
}