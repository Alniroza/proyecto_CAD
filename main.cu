#include <iostream>
#include <cstdio>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

void Read(int** A, int** B, int *la, int *lb, const char *filename) {    
	FILE *fp;
	fp = fopen(filename, "r");
  	fscanf(fp, "%d %d\n", la, lb);

	int* Atemp = new int[(*la) * 2];
	int* Btemp = new int[(*lb) * 2];

	for (int i = 0; i < (*la); i++){
		fscanf(fp, "%d %d\n", &(Atemp[2*i]), &(Atemp[2*i + 1]));
	}

	for (int j = 0; j < (*lb); j++){
		fscanf(fp, "%d %d\n", &(Btemp[2*j]), &(Btemp[2*j + 1]));
	}

	*A = Atemp;
	*B = Btemp;
}

void Write(int* intersecciones, int N, const char *filename) {
	FILE *fp;
	fp = fopen(filename, "w");
	for (int i = 0; i < N; i++){
		if (i%2 == 0){
			fprintf(fp, "%d %d\n", intersecciones[i], intersecciones[i + 1]);
		}
	}
	fclose(fp);
}


bool seInterseca(int aStart, int aEnd, int bStart, int bEnd){

	if ((aEnd < bStart) || (bEnd < aStart)){
		return false;
	}
	else{
		return true;
	}
}

/*
for i en el largo de A
    for j in el largo de B

    Si A[i] se intersecta con B[j]
        Guardar (i,j)
*/
void interseccionConjuntos(int* A, int *B, int *intersecciones,
							int la, int lb){		
	
	int aStart, aEnd, bStart, bEnd;
	int posicion = 0;
	
	for (int i = 0; i < la; i++){
		aStart = A[2*i];
		aEnd = A[2*i + 1];

		for (int j = 0; j < lb; j++){
			bStart = B[2*j];
			bEnd = B[2*j + 1];

			if (seInterseca(aStart, aEnd, bStart, bEnd)){
				//Guardo el número del intervalo (partiendo de 0)
				intersecciones[posicion] = i;
				intersecciones[posicion + 1] = j;
				posicion += 2;
			}
		}
	}
}


//Buscar el indice del intervalo de B que termina antes de que sStart inicie.
//Bend < astart
__device__ void binarySearchEnds(int *B, int lB, int aStart, int *slice){
	int low = 0;
	int high = lB - 1;

	while(low <= high){
		int mid = (low + high)/2;

		if (B[2*mid + 1] >= aStart){
			high = mid - 1;
		} else { // mid > target
			low = mid + 1;
		}
	}
	slice[0] = high;
}

//aend < Bstart
//Deberia ser correcto, buscar el elemento en B, que inicia despues de que sEnd termina.
//O sea, el siguiente numero mayor a sEnd.
__device__ void binarySearchStart(int *B, int lB, int sEnd, int *slice){
	int low = 0;
	int high = lB - 1;

	while(low <= high){
		int mid = (low + high)/2;
		if (B[2*mid] <= sEnd){
			low = mid + 1;
		} else { // mid > target
			high = mid - 1;
		}
	}
	slice[1] = low;
}

__device__ bool isAnIntersect(int aStart, int aEnd, int bStart, int bEnd){
	if ((aEnd < bStart) || (bEnd < aStart)){
		return false;
	}
	else{
		return true;
	}
}



__global__ void setIntersection_Kernel2(int *A, int *B, int lA, int lB, int *intercepts, int *lenIntercepts){
	int Id = threadIdx.x + blockIdx.x * blockDim.x;

	if (Id >= lA) return;

	int* slice = new int[2];
	int aStart = A[2*Id];
	int aEnd = A[2*Id + 1];

	binarySearchEnds(B, lB, aStart, slice);
	binarySearchStart(B, lB, aEnd, slice);

	printf("(%d, %d) - slice (%d, %d)\n",aStart, aEnd, slice[0]+1, slice[1]-1);

	if (slice[0] > slice[1]){
		//No hay interseccion.
		return;
	}

	int *tempInter = new int[2*(slice[1] - slice[0])];  
	int tempInterFounds = 0;
	int bStart, bEnd;
	
	for (int i = slice[0]; i <= slice[1]; i++){
		bStart = B[2*i];
		bEnd = B[2*i + 1];
		if (isAnIntersect(aStart, aEnd, bStart, bEnd)){
			tempInter[2 * tempInterFounds] = Id;
			tempInter[2 * tempInterFounds + 1] = i; 
			tempInterFounds += 1;
		}
	}
	
	__syncthreads();
	atomicAdd(lenIntercepts, tempInterFounds);

}

int main(int argc, char **argv){

	// Largo del arreglo A y B, respectivamente.
	int la, lb;
	// Conjuntos de intervalos A y B.
	int *A, *B;
	int *intersecciones;
	clock_t t1, t2;
	
	char filename[] = {"inputmid.txt\0"};
	char outputFilename[] = {"output.txt\0"};

	Read(&A, &B, &la, &lb, filename); 
	
	// Parte CPU

	intersecciones = new int[la*lb*2];

	t1 = clock();
	interseccionConjuntos(A, B, intersecciones, la, lb);
	t2 = clock();

	double ms = 1000.0 * (double)(t2 -t1) / CLOCKS_PER_SEC;

	std::cout << "Tiempo algoritmo en CPU = " << ms << "[ms]" << std::endl;
	
	int N = sizeof(intersecciones);

	Write(intersecciones, N, outputFilename);

	delete[] intersecciones;

	//Kernel 2 - Binary Search + ...

	hipEvent_t ct1, ct2;
	int *Adev, *Bdev;
	int *interdev, *interhost;
	int *intercepts, *interceptsdev, *lenIntercepts, *lenInterceptsdev;
	lenIntercepts = 0;

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);

    //KERNEL 1

    int gs, bs;
    hipMalloc((void**)&Adev, 2 * la * sizeof(int));
    hipMalloc((void**)&Bdev, 2 * lb * sizeof(int));
    hipMalloc((void**)&interceptsdev, 2 * la * lb * sizeof(int));
    hipMalloc((void**)&lenInterceptsdev, sizeof(int));

    hipMemcpy(Adev, A, 2 * la * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Bdev, B, 2 * lb * sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(lenInterceptsdev, lenIntercepts, sizeof(int), hipMemcpyHostToDevice); 

    bs = 256;
    gs = (int)ceil((float) la / bs);

    hipEventRecord(ct1);
    setIntersection_Kernel2<<<gs, bs>>>(Adev, Bdev, la, lb, interceptsdev, lenInterceptsdev);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);

    float dt;
    hipEventElapsedTime(&dt, ct1, ct2);

    //cudaMemcpy(&maxValueHost, max, sizeof(int), cudaMemcpyDeviceToHost);
    printf("\nTiempo GPU1: %f[ms]\n", dt);
    //printf("Maximo: %d\n", maxValueHost);


}